/**********************************************************************************/
/* MIT License                                                                    */
/*                                                                                */
/* Copyright (c) 2020, 2021 JetBrains-Research                                    */
/*                                                                                */
/* Permission is hereby granted, free of charge, to any person obtaining a copy   */
/* of this software and associated documentation files (the "Software"), to deal  */
/* in the Software without restriction, including without limitation the rights   */
/* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      */
/* copies of the Software, and to permit persons to whom the Software is          */
/* furnished to do so, subject to the following conditions:                       */
/*                                                                                */
/* The above copyright notice and this permission notice shall be included in all */
/* copies or substantial portions of the Software.                                */
/*                                                                                */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     */
/* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       */
/* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    */
/* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         */
/* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  */
/* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  */
/* SOFTWARE.                                                                      */
/**********************************************************************************/

#include <cuda/cuda_backend.hpp>
#include <cuda/cuda_matrix.hpp>
#include <cuda/cuda_vector.hpp>
#include <core/library.hpp>
#include <io/logger.hpp>

#include <iostream>


void cubool::CudaBackend::test() {
    CudaMatrix::test();
}

namespace cubool {

    void CudaBackend::initialize(hints initHints) {
        if (CudaInstance::isCudaDeviceSupported()) {
            mInstance = new CudaInstance(initHints & CUBOOL_HINT_GPU_MEM_MANAGED);
        }

#ifdef CUBOOL_DEBUG
        if (mInstance == nullptr) {
            // No device. Cannot init this backend
            std::cerr << "Failed to initialize Cuda-backend" << std::endl;
        }
#endif
    }

    void CudaBackend::finalize() {
        assert(mMatCount == 0);
        assert(mVecCount == 0);

        if (mMatCount > 0) {
            LogStream stream(*Library::getLogger());
            stream << Logger::Level::Error
                   << "Lost some (" << mMatCount << ") matrix objects" << LogStream::cmt;
        }

        if (mVecCount > 0) {
            LogStream stream(*Library::getLogger());
            stream << Logger::Level::Error
                   << "Lost some (" << mVecCount << ") vector objects" << LogStream::cmt;
        }

        if (mInstance) {
            delete mInstance;
            mInstance = nullptr;
        }
    }

    bool CudaBackend::isInitialized() const {
        return mInstance != nullptr;
    }

    MatrixBase *CudaBackend::createMatrix(size_t nrows, size_t ncols) {
        mMatCount++;
        return new CudaMatrix(nrows, ncols, getInstance());
    }

    VectorBase* CudaBackend::createVector(size_t nrows) {
        mVecCount++;
        return new CudaVector(nrows, getInstance());
    }

    void CudaBackend::releaseMatrix(MatrixBase *matrixBase) {
        mMatCount--;
        delete matrixBase;
    }

    void CudaBackend::releaseVector(VectorBase *vectorBase) {
        mVecCount--;
        delete vectorBase;
    }

    void CudaBackend::queryCapabilities(cuBool_DeviceCaps &caps) {
        mInstance->queryDeviceCapabilities(caps);
    }

    CudaInstance & CudaBackend::getInstance() {
        return *mInstance;
    }

}